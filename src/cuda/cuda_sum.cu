#include "hip/hip_runtime.h"
#include <stdint.h>

#include <hip/hip_runtime.h>

__global__ void kernelSumReduce(uint32_t *g_idata, uint32_t *g_odata) {
    __shared__ uint32_t sdata[256];

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[threadIdx.x] = g_idata[i];

    __syncthreads();
    for (size_t s=1; s < blockDim.x; s *=2)
    {
        int index = 2 * s * threadIdx.x;;

        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(g_odata, sdata[0]);
    }
}

__global__ void
kernelSum(const uint32_t* data, uint32_t* output, size_t work_count, size_t threads) {
    size_t my_id = blockDim.x * blockIdx.x + threadIdx.x;

    uint32_t acc = 0;

    for (size_t i = 0; i < work_count; i++) {
        // acc += data[my_id * work_count + i];
        acc += data[my_id + threads * i];
    }

    output[my_id] = acc;
}

__global__ void
kernelSumFinal(uint32_t* data, size_t size) {
    for (size_t i = 1; i < size; i++) {
        data[0] += data[i];
    }
}
__global__ void
kernelEmpty() {
}

uint32_t* U32_DATA = nullptr;
size_t U32_DATA_SIZE = 0;

uint32_t* U32_OUTPUT_BUFFER = nullptr;
uint32_t* U32_OUTPUT_BUFFER_CPU = nullptr;
size_t U32_OUTPUT_BUFFER_SIZE = 0;


extern  "C" {
    void cuda_empty_kernel() {
        kernelEmpty<<<1, 1>>>();
        hipDeviceSynchronize();
    }

    void cuda_accumulate_u32_free_data() {
        if (U32_DATA != nullptr) {
            hipFree(U32_DATA);
            U32_DATA = nullptr;
            U32_DATA_SIZE = 0;
        }
    }
    void cuda_accumulate_u32_set_data(uint32_t* data, size_t count) {
        cuda_accumulate_u32_free_data();

        hipMalloc(&U32_DATA, sizeof(uint32_t) * count);
        U32_DATA_SIZE = count;

        hipMemcpy(U32_DATA, data, sizeof(uint32_t) * count, hipMemcpyHostToDevice);
    }


    uint32_t cuda_accumulate_u32_sum_subgroup() {
        uint32_t* output;
        hipMalloc(&output, sizeof(uint32_t));
        hipMemset(output, 0, sizeof(uint32_t));

        size_t subgroup_size = min(U32_DATA_SIZE, (size_t) 256);

        kernelSumReduce<<<U32_DATA_SIZE/subgroup_size, subgroup_size>>>(U32_DATA, output);
        hipDeviceSynchronize();

        uint32_t result;
        hipMemcpy(&result, output, sizeof(uint32_t), hipMemcpyDeviceToHost);

        hipFree(output);
        return result;
    }

    uint32_t cuda_accumulate_u32_sum(
            size_t total_threads,
            size_t subgroup_size,

            size_t second_accumulate_on_gpu
    ) {
        if (U32_OUTPUT_BUFFER_SIZE != total_threads) {
            if (U32_OUTPUT_BUFFER) {
                hipFree(U32_OUTPUT_BUFFER);
                free(U32_OUTPUT_BUFFER_CPU);
            }

            hipMalloc(&U32_OUTPUT_BUFFER, sizeof(uint32_t) * total_threads);
            U32_OUTPUT_BUFFER_CPU = (uint32_t*) malloc(sizeof(uint32_t) * total_threads);
            U32_OUTPUT_BUFFER_SIZE = total_threads;
        }

        kernelSum<<<total_threads / subgroup_size, subgroup_size>>>(U32_DATA, U32_OUTPUT_BUFFER, U32_DATA_SIZE / total_threads, total_threads);
        if (second_accumulate_on_gpu) {
            kernelSumFinal<<<1, 1>>>(U32_OUTPUT_BUFFER, total_threads);
        }
        hipDeviceSynchronize();

        hipMemcpy(
                U32_OUTPUT_BUFFER_CPU,
                U32_OUTPUT_BUFFER,
                sizeof(uint32_t) * (second_accumulate_on_gpu ? second_accumulate_on_gpu : total_threads),
                hipMemcpyDeviceToHost
        );

        if (second_accumulate_on_gpu != 1) {
            if (second_accumulate_on_gpu) {
                for (size_t i = 1; i < second_accumulate_on_gpu; i++) {
                    U32_OUTPUT_BUFFER_CPU[0] += U32_OUTPUT_BUFFER_CPU[i];
                }
            } else {
                for (size_t i = 1; i < total_threads; i++) {
                    U32_OUTPUT_BUFFER_CPU[0] += U32_OUTPUT_BUFFER_CPU[i];
                }
            }
        }
        uint32_t result = U32_OUTPUT_BUFFER_CPU[0];

        return result;
    }

}
