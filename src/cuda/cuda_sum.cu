#include "hip/hip_runtime.h"
#include <stdint.h>

#include <hip/hip_runtime.h>

__global__ void kernelSumReduce(uint32_t *g_idata, uint32_t *g_odata) {
    __shared__ uint32_t sdata[256];

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[threadIdx.x] = g_idata[i];

    __syncthreads();
    for (size_t s=1; s < blockDim.x; s *=2)
    {
        int index = 2 * s * threadIdx.x;;

        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(g_odata, sdata[0]);
    }
}

__global__ void
kernelSum(const uint32_t* data, uint32_t* output, size_t work_count, size_t threads) {
    size_t my_id = blockDim.x * blockIdx.x + threadIdx.x;

    uint32_t acc = 0;

    for (size_t i = 0; i < work_count; i++) {
        // acc += data[my_id * work_count + i];
        acc += data[my_id + threads * i];
    }

    output[my_id] = acc;
}

__global__ void
kernelSumFinal(uint32_t* data, size_t size) {
    for (size_t i = 1; i < size; i++) {
        data[0] += data[i];
    }
}
__global__ void
kernelEmpty() {
}

uint32_t* U32_DATA = nullptr;
size_t U32_DATA_SIZE = 0;

uint32_t* U32_OUTPUT_BUFFER = nullptr;
size_t U32_OUTPUT_BUFFER_SIZE = 0;


extern  "C" {
    void cuda_empty_kernel() {
        kernelEmpty<<<1, 1>>>();
        hipDeviceSynchronize();
    }

    void cuda_accumulate_u32_free_data() {
        if (U32_DATA != nullptr) {
            hipFree(U32_DATA);
            U32_DATA = nullptr;
            U32_DATA_SIZE = 0;
        }
    }
    void cuda_accumulate_u32_set_data(uint32_t* data, size_t count) {
        cuda_accumulate_u32_free_data();

        hipMallocManaged(&U32_DATA, sizeof(uint32_t) * count);
        U32_DATA_SIZE = count;

        memcpy(U32_DATA, data, sizeof(uint32_t) * count);
    }


    uint32_t cuda_accumulate_u32_sum__(
            size_t total_threads,
            size_t subgroup_size,

            size_t second_accumulate_on_gpu
    ) {
        uint32_t* output;
        hipMallocManaged(&output, sizeof(uint32_t));
        *output = 0;

        subgroup_size = min(subgroup_size, (size_t) 256);

        kernelSumReduce<<<U32_DATA_SIZE/subgroup_size, subgroup_size>>>(U32_DATA, output);
        hipDeviceSynchronize();

        uint32_t result = *output;
        hipFree(output);
        return result;
    }

    uint32_t cuda_accumulate_u32_sum(
            size_t total_threads,
            size_t subgroup_size,

            size_t second_accumulate_on_gpu
    ) {
        if (U32_OUTPUT_BUFFER_SIZE != total_threads) {
            if (U32_OUTPUT_BUFFER) {
                hipFree(U32_OUTPUT_BUFFER);
            }

            hipMallocManaged(&U32_OUTPUT_BUFFER, sizeof(uint32_t) * total_threads);
            U32_OUTPUT_BUFFER_SIZE = total_threads;
        }

        kernelSum<<<total_threads / subgroup_size, subgroup_size>>>(U32_DATA, U32_OUTPUT_BUFFER, U32_DATA_SIZE / total_threads, total_threads);
        if (second_accumulate_on_gpu) {
            kernelSumFinal<<<1, 1>>>(U32_OUTPUT_BUFFER, total_threads);
        }
        hipDeviceSynchronize();

        if (second_accumulate_on_gpu != 1) {
            if (second_accumulate_on_gpu) {
                for (size_t i = 1; i < second_accumulate_on_gpu; i++) {
                    U32_OUTPUT_BUFFER[0] += U32_OUTPUT_BUFFER[i];
                }
            } else {
                for (size_t i = 1; i < total_threads; i++) {
                    U32_OUTPUT_BUFFER[0] += U32_OUTPUT_BUFFER[i];
                }
            }
        }
        uint32_t result = U32_OUTPUT_BUFFER[0];

        return result;
    }
}
